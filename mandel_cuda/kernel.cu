
#include "hip/hip_runtime.h"

#include "hip/hip_complex.h"
#include "math.h"

__device__ double map(double old_value, double old_min, double old_max, double new_min, double new_max)
{
	double old_range = old_max - old_min;
	double new_range = new_max - new_min;

	return (((old_value - old_min) * new_range) / old_range) + new_min;
}

__device__ int lerp_colors(int color1, int color2, double v)
{
	int red1 = (color1 >> 16) & 0xFF;
	int green1 = (color1 >> 8) & 0xFF;
	int blue1 = color1 & 0xFF;

	int red2 = (color2 >> 16) & 0xFF;
	int green2 = (color2 >> 8) & 0xFF;
	int blue2 = color2 & 0xFF;

	double inverse_v = 1 - v;

	int redPart = (int)(red1 * inverse_v + red2 * v);
	int greenPart = (int)(green1 * inverse_v + green2 * v);
	int bluePart = (int)(blue1 * inverse_v + blue2 * v);

	return (255 << 24) + (redPart << 16) + (greenPart << 8) + (bluePart);
}

__device__ int color_from_iter(int iterCount, double znMagn, int *palette, int paletteLength) {
	double temp_i = iterCount;

	// sqrt of inner term removed using log simplification rules.
	double log_zn = log2(znMagn) / 2;
	double nu = log2(log_zn / log2(2.0)) / log2(2.0);
	// Rearranging the potential function.
	// Dividing log_zn by log(2) instead of log(N = 1<<8)
	// because we want the entire palette to range from the
	// center to radius 2, NOT our bailout radius.
	temp_i = temp_i + 1 - nu;
	// Grab two colors from the pallete
	int color1 = palette[(int)temp_i % (paletteLength - 1)];
	int color2 = palette[(int)(temp_i + 1) % (paletteLength - 1)];

	// Lerp between both colors
	int final = lerp_colors(color1, color2, fmod(temp_i, 1.0));

	// Return the result.
	return final;
}

extern "C" {
	__global__ void perturbation(
		int *out, int *palette, int paletteLength,
		hipDoubleComplex *points, int pointCount,
		int cell_x, int cell_y,
		int cellWidth, int cellHeight,
		int totalCells_x, int totalCells_y,
		double xMax, double yMax,
		int chunkSize, int maxChunkSize) {

		unsigned int x_dim = cell_x * cellWidth + blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y_dim = cell_y * cellHeight + blockIdx.y * blockDim.y + threadIdx.y;

		if (x_dim % chunkSize != 0 || y_dim % chunkSize != 0)
			return;

		unsigned int frameWidth = cellWidth * totalCells_x;
		unsigned int frameHeight = cellHeight * totalCells_y;

		int index = frameWidth * y_dim + x_dim;


		double x_origin = map(x_dim, 0, frameWidth, -xMax, xMax);
		double y_origin = map(y_dim, 0, frameHeight, -yMax, yMax);

		int iter = 0;

		int max_iter = pointCount;

		// Initialize some variables...
		hipDoubleComplex zn;

		hipDoubleComplex d0 = make_hipDoubleComplex(x_origin, y_origin);

		hipDoubleComplex dn = d0;

		double zn_r = 0;
		double zn_i = 0;

		double zn_magn = 0;

		// Mandelbrot algorithm
		if ((x_dim / chunkSize) % 2 == 0 && (y_dim / chunkSize) % 2 == 0 && chunkSize != maxChunkSize)
			return;
		do
		{
			// dn *= iter_list[iter] + dn
			dn = hipCmul(dn, hipCadd(points[iter], dn));

			// dn += d0
			dn = hipCadd(dn, d0);

			iter++;

			// zn = x[iter] * 0.5 + dn
			zn = hipCadd(hipCmul(points[iter], make_hipDoubleComplex(0.5, 0)), dn);

			zn_r = hipCreal(zn);
			zn_i = hipCimag(zn);

			zn_magn = zn_r * zn_r + zn_i * zn_i;

		} while (zn_magn < 256 && iter < max_iter);

		int color = 0;
		if (iter == max_iter) {
			color = 255 << 24;
		}
		else {
			color = color_from_iter(iter, zn_magn, palette, paletteLength);
		}
		for (int j = y_dim; j < y_dim + chunkSize; j++)
		{
			for (int i = x_dim; i < x_dim + chunkSize; i++)
			{
				int index = i + j * frameWidth;
				if(index < frameWidth * frameHeight)
					out[i + j * frameWidth] = color;
			}
		}
	}

	__global__ void traditional(
		int *out, int *palette,
		int paletteLength,
		int cell_x, int cell_y,
		int cellWidth, int cellHeight,
		int totalCells_x, int totalCells_y,
		double xMax, double yMax,
		double offset_x, double offset_y,
		int max_iteration, 
		int chunkSize, int maxChunkSize) {

		unsigned int x_dim = cell_x * cellWidth + blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y_dim = cell_y * cellHeight + blockIdx.y * blockDim.y + threadIdx.y;

		if (x_dim % chunkSize != 0 || y_dim % chunkSize != 0)
			return;

		unsigned int frameWidth = cellWidth * totalCells_x;
		unsigned int frameHeight = cellHeight * totalCells_y;

		double x_origin = map(x_dim, 0, frameWidth, -xMax, xMax) + offset_x;
		double y_origin = map(y_dim, 0, frameHeight, -yMax, yMax) + offset_y;

		double x = 0.0;
		double y = 0.0;

		double xx = 0.0;
		double yy = 0.0;

		int iteration = 0;

		if ((x_dim / chunkSize) % 2 == 0 && (y_dim / chunkSize) % 2 == 0 && chunkSize != maxChunkSize)
			return;

		while (xx + yy <= 4 && iteration < max_iteration) {
			double xtemp = xx - yy + x_origin;
			double ytemp = 2 * x * y + y_origin;

			if (x == xtemp && y == ytemp)
			{
				iteration = max_iteration;
				break;
			}

			x = xtemp;
			y = ytemp;

			xx = x * x;
			yy = y * y;

			iteration++;
		}
		int color = 0;
		if (iteration == max_iteration) {
			color = 255 << 24;
		}
		else {
			color = color_from_iter(iteration, xx + yy, palette, paletteLength);
		}
		for (int j = y_dim; j < y_dim + chunkSize; j++)
		{
			for (int i = x_dim; i < x_dim + chunkSize; i++)
			{
				int index = i + j * frameWidth;
				if(index < frameWidth * frameHeight)
					out[index] = color;
			}
		}
	}
}