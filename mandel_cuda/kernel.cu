
#include "hip/hip_runtime.h"

#include "hip/hip_complex.h"
#include "math.h"

extern "C" {
	__device__ double map(double old_value, double old_min, double old_max, double new_min, double new_max)
	{
		double old_range = old_max - old_min;
		double new_range = new_max - new_min;

		return (((old_value - old_min) * new_range) / old_range) + new_min;
	}

	__device__ int lerp_colors(int color1, int color2, double v) 
	{
		int red1 = (color1 >> 16) & 0xFF;
		int green1 = (color1 >> 8) & 0xFF;
		int blue1 = color1 & 0xFF;

		int red2 = (color2 >> 16) & 0xFF;
		int green2 = (color2 >> 8) & 0xFF;
		int blue2 = color2 & 0xFF;

		double inverse_v = 1 - v;

		int redPart = (int)(red1 * inverse_v + red2 * v);
		int greenPart = (int)(green1 * inverse_v + green2 * v);
		int bluePart = (int)(blue1 * inverse_v + blue2 * v);

		return (255 << 24) + (redPart << 16) + (greenPart << 8) + (bluePart);
	}

	__device__ int color_from_iter(int iterCount, double znMagn, int *palette, int paletteLength) {
		double temp_i = iterCount;
		
		// sqrt of inner term removed using log simplification rules.
		double log_zn = log2(znMagn) / 2;
		double nu = log2(log_zn / log2(2.0)) / log2(2.0);
		// Rearranging the potential function.
		// Dividing log_zn by log(2) instead of log(N = 1<<8)
		// because we want the entire palette to range from the
		// center to radius 2, NOT our bailout radius.
		temp_i = temp_i + 1 - nu;
		// Grab two colors from the pallete
		int color1 = palette[(int)temp_i % (paletteLength - 1)];
		int color2 = palette[(int)(temp_i + 1) % (paletteLength - 1)];

		// Lerp between both colors
		int final = lerp_colors(color1, color2, fmod(temp_i, 1.0));

		// Return the result.
		return final;
	}

	__global__ void render(int *out, int *palette, int paletteLength, int width, int height, double offset_x, double offset_y, double zoom, int max_iteration) {
		unsigned int x_dim = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y_dim = blockIdx.y*blockDim.y + threadIdx.y;
		int index = width * y_dim + x_dim;

		double scaleFactor = ((double)width / (double)height) * 2;


		double x_origin = map(x_dim, 0, width, -scaleFactor / zoom, scaleFactor / zoom) + offset_x;
		double y_origin = map(y_dim, 0, height, -2 / zoom, 2 / zoom) + offset_y;

		double x = 0.0;
		double y = 0.0;

		int iteration = 0;
		while (x*x + y * y <= 4 && iteration < max_iteration) {
			double xtemp = x * x - y * y + x_origin;
			y = 2 * x*y + y_origin;
			x = xtemp;
			iteration++;
		}

		if (iteration == max_iteration) {
			out[index] = 0;
		}
		else {
			out[index] = color_from_iter(iteration, x * x + y * y, palette, paletteLength);
		}
	}
}