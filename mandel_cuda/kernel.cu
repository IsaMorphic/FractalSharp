
#include "hip/hip_runtime.h"

#include "hip/hip_complex.h"
#include "math.h"

extern "C" {
	__device__ double map(double old_value, double old_min, double old_max, double new_min, double new_max)
	{
		double old_range = old_max - old_min;
		double new_range = new_max - new_min;

		return (((old_value - old_min) * new_range) / old_range) + new_min;
	}

	__device__ int lerp_colors(int color1, int color2, double v) 
	{
		int red1 = (color1 >> 16) & 0xFF;
		int green1 = (color1 >> 8) & 0xFF;
		int blue1 = color1 & 0xFF;

		int red2 = (color2 >> 16) & 0xFF;
		int green2 = (color2 >> 8) & 0xFF;
		int blue2 = color2 & 0xFF;

		double inverse_v = 1 - v;

		int redPart = (int)(red1 * inverse_v + red2 * v);
		int greenPart = (int)(green1 * inverse_v + green2 * v);
		int bluePart = (int)(blue1 * inverse_v + blue2 * v);

		return (255 << 24) + (redPart << 16) + (greenPart << 8) + (bluePart);
	}

	__device__ int color_from_iter(int iterCount, double znMagn, int *palette, int paletteLength) {
		double temp_i = iterCount;
		
		// sqrt of inner term removed using log simplification rules.
		double log_zn = log2(znMagn) / 2;
		double nu = log2(log_zn / log2(2.0)) / log2(2.0);
		// Rearranging the potential function.
		// Dividing log_zn by log(2) instead of log(N = 1<<8)
		// because we want the entire palette to range from the
		// center to radius 2, NOT our bailout radius.
		temp_i = temp_i + 1 - nu;
		// Grab two colors from the pallete
		int color1 = palette[(int)temp_i % (paletteLength - 1)];
		int color2 = palette[(int)(temp_i + 1) % (paletteLength - 1)];

		// Lerp between both colors
		int final = lerp_colors(color1, color2, fmod(temp_i, 1.0));

		// Return the result.
		return final;
	}

	__global__ void get_points(hipDoubleComplex *points_out, int *pointsCount, double offset_x, double offset_y, int max_iter) 
	{
		double xn_r = offset_x;
		double xn_i = offset_y;

		int i;

		for (i = 0; i < max_iter; i++) 
		{
			double real = xn_r + xn_r;
			double imag = xn_i + xn_i;

			double xn_r2 = xn_r * xn_r;
			double xn_i2 = xn_i * xn_i;

			points_out[i] = make_hipDoubleComplex(real, imag);

			if (real > 1024 || real < -1024 ||
				imag > 1024 || imag < -1024)
				break;

			xn_r = xn_r2 - xn_i2 + offset_x;
			xn_i = real * xn_i + offset_y;
		}

		*pointsCount = i;
	}

	__global__ void perturbation(int *out, int *palette, int paletteLength, hipDoubleComplex *points, int pointCount, int width, int height, double xMax, double yMax) {
		unsigned int x_dim = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y_dim = blockIdx.y*blockDim.y + threadIdx.y;
		int index = width * y_dim + x_dim;


		double x_origin = map(x_dim, 0, width, -xMax, xMax);
		double y_origin = map(y_dim, 0, height, -yMax, yMax);

		int iter = 0;
		
		int max_iter = pointCount;

		// Initialize some variables...
		hipDoubleComplex zn;

		hipDoubleComplex d0 = make_hipDoubleComplex(x_origin, y_origin);

		hipDoubleComplex dn = d0;

		double zn_r = 0;
		double zn_i = 0;

		double zn_magn = 0;

		// Mandelbrot algorithm
		do
		{
			// dn *= iter_list[iter] + dn
			dn = hipCmul(dn, hipCadd(points[iter], dn));

			// dn += d0
			dn = hipCadd(dn, d0);

			iter++;

			// zn = x[iter] * 0.5 + dn
			zn = hipCadd(hipCmul(points[iter], make_hipDoubleComplex(0.5, 0)), dn);

			zn_r = hipCreal(zn);
			zn_i = hipCimag(zn);

			zn_magn = zn_r * zn_r + zn_i * zn_i;

		} while (zn_magn < 256 && iter < max_iter);

		if (iter == max_iter) {
			out[index] = 255 << 24;
		}
		else {
			out[index] = color_from_iter(iter, zn_magn, palette, paletteLength);
		}
	}

	__global__ void traditional(
		int *out, int *palette, int paletteLength, 
		int cell_x, int cell_y, 
		int cellWidth, int cellHeight, 
		int frameWidth, int frameHeight, 
		double xMax, double yMax, 
		double offset_x, double offset_y, 
		int max_iteration) {

		unsigned int x_dim = cell_x * cellWidth + blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int y_dim = cell_y * cellHeight + blockIdx.y*blockDim.y + threadIdx.y;
		int index = frameWidth * y_dim + x_dim;

		double x_origin = map(x_dim, 0, frameWidth, -xMax, xMax) + offset_x;
		double y_origin = map(y_dim, 0, frameHeight, -yMax, yMax) + offset_y;

		double x = 0.0;
		double y = 0.0;

		double xx = 0.0;
		double yy = 0.0;

		int iteration = 0;
		while (xx + yy <= 4 && iteration < max_iteration) {
			double xtemp = xx - yy + x_origin;
			double ytemp = 2 * x * y + y_origin;

			if (x == xtemp && y == ytemp) 
			{
				iteration = max_iteration;
				break;
			}

			x = xtemp;
			y = ytemp;

			xx = x * x;
			yy = y * y;

			iteration++;
		}

		if (iteration == max_iteration) {
			out[index] = 255 << 24;
		}
		else {
			out[index] = color_from_iter(iteration, xx + yy, palette, paletteLength);
		}
	}
}